/* *********************************************************************
* DESCRIPTION: 
* Serial Concurrent Wave Equation - C Version 
* This program implements the concurrent wave equation 
*********************************************************************/ 

#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include <time.h> 
#define MAXPOINTS 1000000 
#define MAXSTEPS 1000000 
#define MINPOINTS 20 
#define PI 3.14159265 

void check_param(void); 
void printfinal (void); 


int nsteps ,/* number of time steps */ 
    tpoints ,/* total points along string */ 
    rcode;/* generic return code */ 




/* *********************************************************************
* Checks input values from parameters 
*********************************************************************/ 
void check_param(void) 
{ 
    char tchar[20]; 
    /* check number of points , number of iterations */ 
    while (( tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) { 
        printf("Enter number of points along vibrating string [%d-%d]: " ,MINPOINTS, MAXPOINTS); 
        scanf("%s", tchar); 
        tpoints = atoi(tchar); 
        if (( tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) 
            printf("Invalid. Please enter value between %d and %d\n", MINPOINTS , MAXPOINTS); 
    } 
    while (( nsteps < 1) || (nsteps > MAXSTEPS)) { 
            printf("Enter number of time steps [1-%d]: ", MAXSTEPS); 
            scanf("%s", tchar); 
            nsteps = atoi(tchar); 
            if ((nsteps < 1) || (nsteps > MAXSTEPS)) 
                printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS); 
    } 
    printf("Using points = %d, steps = %d\n", tpoints , nsteps); 
    
} 
/* ********************************************************************** 
Initialize points on line 
*********************************************************************/ 
__global__ void init_line_kernel(float* oldval, float* values, int tpoints) { 
    int j; 
    float x, fac , k, tmp; 
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_num = blockDim.x * gridDim.x;
    
    /* Calculate initial values based on sine curve */ 
    fac = 2.0 * PI; 
    tmp = (float)(tpoints - 1); 
    for (j = thread_id; j <= tpoints; j += thread_num) { 
        if (j >= 0){
            k = (float)(j - 1);
            x = k/tmp; 
            values[j] = __sinf(fac * x); //GPU內建  __sinf
            oldval[j] = values[j]; 
        }

    } 
}

/* ********************************************************************** 
Update all values along line a specified number of times 
*********************************************************************/ 
__global__ void update_kernel(float* oldval, float* values, float* newval, int nsteps, int tpoints){
    int i, j; 
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_num = blockDim.x * gridDim.x;
    /* Update values for each time step */ 
    for (i = 1; i <= nsteps; i++) { 
    /* Update points along line for this time step */ 
        for (j = thread_id; j <= tpoints; j += thread_num) { 
            /* global endpoints */ 
            if ((j == 1) || (j == tpoints)){
                newval[j] = 0.0; 
            }else {
                /*********************************************************************** 
                Calculate new values using wave equation 
                *********************************************************************/ 
                float dtime , c, dx, tau , sqtau; 
                dtime = 0.3; 
                c = 1.0; 
                dx = 1.0; 
                tau = (c * dtime / dx); 
                sqtau = tau * tau; 
                newval[j] = (2.0 * values[j]) - oldval[j] + (sqtau * ( -2.0)*values[j]); 
            }
            oldval[j] = values[j]; 
            values[j] = newval[j]; 
        } 
    }
} 
/* ********************************************************************** 
Print final results 
*********************************************************************/ 
void printfinal (float* values, int tpoints) { 
    int i; 
    for (i = 1; i <= tpoints; i++) { 
        printf("%6.4f ", values[i]); 
        if (i%10 == 0) 
            printf("\n"); 
    } 
} 
/* ********************************************************************** 
Main program 
*********************************************************************/ 
int main(int argc , char *argv []) { 
    sscanf(argv[1],"%d" ,&tpoints); 
    sscanf(argv[2],"%d" ,&nsteps); 
    check_param(); 
    int threadinblock = 512;
    int blocknum = (tpoints + threadinblock - 1)/threadinblock; // 無條件補上
    //Host
    float *oldval, *values, *newval;
    oldval = (float*)malloc((tpoints + 2) * sizeof(float)); /* values at time (t-dt) */
    values = (float*)malloc((tpoints + 2) * sizeof(float)); /* values at time t */
    newval = (float*)malloc((tpoints + 2) * sizeof(float)); /* values at time (t+dt) */
    
    //Device
    float *gpu_oldval, *gpu_values, *gpu_newval;
    hipMalloc(&gpu_oldval, (tpoints + 2) * sizeof(float));
    hipMalloc(&gpu_values, (tpoints + 2) * sizeof(float));
    hipMalloc(&gpu_newval, (tpoints + 2) * sizeof(float));



    printf("Initializing points on the line ...\n"); 
    init_line_kernel <<<blocknum, threadinblock>>> (gpu_oldval, gpu_values, tpoints);

    printf("Updating all points for all time steps ...\n"); 
    update_kernel <<<blocknum, threadinblock>>> (gpu_oldval, gpu_values, gpu_newval, nsteps, tpoints);

    //把資料送回Host
    hipMemcpy(oldval, gpu_oldval, (tpoints + 2) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(values, gpu_values, (tpoints + 2) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(newval, gpu_newval, (tpoints + 2) * sizeof(float), hipMemcpyDeviceToHost);

    printf("Printing final results ...\n"); 
    printfinal(values, tpoints); 
    printf("\nDone .\n\n"); 
    return 0; 
} 